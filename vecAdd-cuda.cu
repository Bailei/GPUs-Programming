#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define W 100
#define THREAD_NUM 512
#define BLOCK_NUM 32

bool InitCUDA(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void vector_gen(float* a, int size){
	for(int i = 0; i < size; i++){
		a[i] = rand();
	}
}

__global__ void vecAddkernel(float* A_d, float* B_d, float* C_d, int n, clock_t* time){
	clock_t start = clock();
	int tx = threadIdx.x;
	int i = tx;
	if(i < n) C_d[i] = A_d[i] + B_d[i];
	*time = clock() - start;
}

int main(){
	if(!InitCUDA()){
		return 0;
	}
	printf("CUDA initialized.\n");

	float* A, *B, *C;
	A = (float*) malloc(sizeof(int) * W);
	B = (float*) malloc(sizeof(int) * W);
	C = (float*) malloc(sizeof(int) * W);
	vector_gen(A, W);
	vector_gen(B, W);
	
	int size = W * sizeof(float);
	float* A_d, *B_d, *C_d;
	clock_t *time;
	clock_t time_used;

	hipMalloc((void**) &A_d, size);
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &B_d, size);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &time, sizeof(clock_t));

	hipMalloc((void**) &C_d, size);

	vecAddkernel<<<1, W>>>(A_d, B_d, C_d, W, time);

	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	for(int i = 0; i < W ; i++){
        if(i % 10  == 0){
            printf("\n");
        }
        printf("%f ", C[i]);
	}
	printf("\n");
	printf("time: %ldms\n", time_used);
	return 0;
}