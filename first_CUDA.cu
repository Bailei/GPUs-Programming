#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define DATA_SIZE 999999

int data[DATA_SIZE];

bool InitCUDA(){
	int count;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stdeer, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stdeer, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void GenerateNumber(int *number, int size){
	for(int i = 0; i < size; i++){
		number[i] = rand() % 10;
	}
}

_global_static void sumOfSquares(int *num, int *result， clock_t *time){
	int sum = 0;
	int i;
	for(i = 0; i < DATA_SIZE; i++){
		sum += num[i]*num[i];
	}
	*result = sum;
	*time = clock() - start;
}


int main(){
	if(!InitCUDA()){
		return 0;
	}	

	printf("CUDA initialized.\n");

	GenerateNumber(data, DATA_SIZE);

	int* gpudata, int* result;
	cudaMallo((void**) &gpudata, sizeof(int)*DATA_SIZE);
	cudaMallo((void**) &result, sizeof(int));
	hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);

	
	sumofSquares<<<1, 1, 0>>>(gpudata, result, time);
	int sum;
	clock_t time_used;
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, sizeof(clock_t), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);

	printf("sum: %d time: %d\n", sum, time_used);

	return 0;
}


