#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define DATA_SIZE 999999

int data[DATA_SIZE];

bool InitCUDA(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void GenerateNumber(int* number, int size){
	for(int i = 0; i < size; i++){
		number[i] = rand() % 10;
	}
}

__global__ void sumOfSquares(int* num, int* result, clock_t* time)
{
	int sum = 0;
	int i;
	clock_t start = clock();
	for(i = 0; i < DATA_SIZE; i++){
		sum += num[i] * num[i];
	}
	*result = sum;
	*time = clock() - start;
}


int main(){
	if(!InitCUDA()){
		return 0;
	}	

	printf("CUDA initialized.\n");

	GenerateNumber(data, DATA_SIZE);

	int* gpudata, *result;
	clock_t* time;
	hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
	hipMalloc((void**) &result, sizeof(int));
	hipMalloc((void**) &time, sizeof(clock_t));
	hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);

	
	sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);
	int sum;
	clock_t time_used;
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);

	printf("sum: %d time: %d\n", sum, time_used);

	return 0;
}


