#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define X 16
#define THREAD_NUM 512
#define BLOCK_NUM 32

bool InitCUDA(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;
}

void matgen(float* a, int n){
	int i, j;
	for(i = 0; i < n; i++){
		for(j = 0; j < n; j++){
			a[i * n + j] = (float)rand();
		}
	}
}	
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd){
	
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float Pvalue = 0;

	for(int k = 0; k < X; k++){
		float Mdelement = Md[tx * X + k];
		float Ndelement = Nd[k * X + ty];
		Pvalue += Mdelement * Ndelement;
	}
	Pd[ty * X + tx] = Pvalue;
}


int main(){
	if(!InitCUDA())
		return 0;
	printf("CUDA initialized.\n");

	clock_t start = clock();
	float* M, *N, *P;
    M = (float*) malloc(sizeof(float) * X * X);
    N = (float*) malloc(sizeof(float) * X * X);
    P = (float*) malloc(sizeof(float) * X * X);
	
	srand(0);
	matgen(M, X);
	matgen(N, X);

	int size = X * X * sizeof(float);
	float* Md, *Nd, *Pd;

	hipMalloc((void**) &Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Pd, size);

	dim3 dimBlock(X, X);
	dim3 dimGrid(1, 1);

	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	clock_t end = clock() - start;

	for(int i = 0; i < X * X; i++){
        if(i % X  == 0){
            //printf("\n");
        }
        //printf("%f ", P[i]);
	}
	printf("\n");
	printf("time: %ldms", end);

	return 0;
}