#include <stdio.h>
#include <hip/hip_runtime.h>

bool InitCUDA(){
	int count;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++){
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	} 

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
	printf("clockRate: %ld\n", prop.clockRate);
	printf("totalGlobalMem: %ld\n", prop.totalGlobalMem);

	return true;
}

int main(){
	InitCUDA();
	printf("CUDA initialized.\n");

	return 0;
}